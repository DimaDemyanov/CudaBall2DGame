#include "hip/hip_runtime.h"
#pragma once
#include"hip/hip_runtime.h"
#include""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include"Ball.h"
#include"math.h"
#include<iostream>
#define gravity 0.2f

hipError_t allocateOnGPU(void** arrayData, int size_in_bytes)
{
  hipError_t cudaStatus = hipMalloc((void**)arrayData, size_in_bytes);

  if (cudaStatus != hipSuccess)
  {
    std::cout << "CudaMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

hipError_t moveToGPU(void* dataCPU, void* dataGPU, int size_in_bytes)
{
  hipError_t cudaStatus = hipMemcpy(dataGPU, dataCPU, size_in_bytes, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    std::cout << "TransferData from CPU to GPU failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

hipError_t moveToCPU(void* dataCPU, void* dataGPU, int size_in_bytes)
{
  hipError_t cudaStatus = hipMemcpy(dataCPU, dataGPU, size_in_bytes, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    std::cout << "TransferData from GPU to CPU failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

__global__ void CudaMoveBallsKernel(BallData* ballsData, int size, float deltaTime)
{
  int tId = blockIdx.x * blockDim.x + threadIdx.x;
  if (tId < size) {
    ballsData[tId].pos_x = ballsData[tId].pos_x - ballsData[tId].velocity_x * deltaTime;
    ballsData[tId].pos_y = ballsData[tId].pos_y - ballsData[tId].velocity_y * deltaTime;
    ballsData[tId].velocity_y += gravity * deltaTime;
  }
}

hipError_t CudaMoveBallsKernelInvoke(BallData* ballsData, int size, float deltaTime)
{
  int threadsPerBlock = 1024;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  CudaMoveBallsKernel<<<blocksPerGrid, threadsPerBlock>>>(ballsData, size, deltaTime);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    std::cout << "CudaMoveBallsKernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

// Resolving collisions between balls
__device__ void CudaResolveBallsCollision(BallData* ball1, BallData* ball2) {
  // get the mtd
  float delta_x = ball1->pos_x - ball2->pos_x;
  float delta_y = ball1->pos_y - ball2->pos_y;

  float d = sqrt(delta_x * delta_x + delta_y * delta_y);
  
  // minimum translation distance to push balls apart after intersecting
  float mtd_x = delta_x * (((ball1->radius + ball2->radius) - d) / d);
  float mtd_y = delta_y * (((ball1->radius + ball2->radius) - d) / d);
  
  // resolve intersection --
  // inverse mass quantities
  float im1 = 1 / ball1->mass;
  float im2 = 1 / ball2->mass;

  // impact speed
  float v_x = ball1->velocity_x - ball2->velocity_x;
  float v_y = ball1->velocity_y - ball2->velocity_y;

  float mtd_len = sqrt(mtd_x * mtd_x + mtd_y * mtd_y);
  float vn = v_x * mtd_x / mtd_len + v_y * mtd_y / mtd_len;

  // sphere intersecting but moving away from each other already
  if (vn < 0.0f) {
    return;
  }

  // collision impulse
  float i = (-(1.0f + 1.0f) * vn) / (im1 + im2);
  float impulse_x = mtd_x / mtd_len * i;
  float impulse_y = mtd_y / mtd_len * i;

  float elastic_coef = 0.5f;
  if (ball1->type == ball2->type) {
    elastic_coef = 0.5f;
  }

  // change in momentum
  ball1->velocity_x = ball1->velocity_x + impulse_x * im1 * elastic_coef;
  ball1->velocity_y = ball1->velocity_y + impulse_y * im1 * elastic_coef;

  ball2->velocity_x = ball2->velocity_x - impulse_x * im2 * elastic_coef;
  ball2->velocity_y = ball2->velocity_y - impulse_y * im2 * elastic_coef;
}

__global__ void CudaResolveBallsCollisionsKernel(BallData* ballsData, int size)
{
  int tId = blockIdx.x * blockDim.x + threadIdx.x;
  if (tId < size) {
    for (int i = tId + 1; i < size; i++) {
      float dist = sqrt((ballsData[tId].pos_x - ballsData[i].pos_x) * (ballsData[tId].pos_x - ballsData[i].pos_x) + (ballsData[tId].pos_y - ballsData[i].pos_y) * (ballsData[tId].pos_y - ballsData[i].pos_y));
      if (ballsData[tId].radius + ballsData[i].radius > dist) {
        CudaResolveBallsCollision(ballsData + tId, ballsData + i);
      }
    }
  }
  __syncthreads();
}

hipError_t CudaResolveBallsCollisionsKernelInvoke(BallData* ballsData, int size)
{
  int threadsPerBlock = 1024;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  CudaResolveBallsCollisionsKernel <<<blocksPerGrid, threadsPerBlock>>> (ballsData, size);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    std::cout << "CudaResolveBallsCollisionsKernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

// Resolving collisions with Platform

__device__ void changeVelocity(float cx, float cy, float radius, float left, float top, float right, float bottom, float velocity_x, float velocity_y, float* newVelocity_x, float* newVelocity_y) {
  float closestX = (cx < left ? left : (cx > right ? right : cx));
  float closestY = (cy < top ? top : (cy > bottom ? bottom : cy));
  float dx = closestX - cx;
  float dy = closestY - cy;

  *newVelocity_x = velocity_x;
  *newVelocity_y = velocity_y;
  if (dy > dx && dx != 0) {
    if (dx * velocity_x > 0) {
      return;
    }
    *newVelocity_x = -velocity_x;
  }
  else {
    if (dy * velocity_y > 0) {
      return;
    }
    *newVelocity_y = -velocity_y;
  }
}

__device__ void changeVelocity(float cx, float cy, float radius, float x, float y, float width, float height, float angle, float velocity_x, float velocity_y, float* newVelocity_x, float* newVelocity_y) {
  float alpha = -angle / 180 * PI;
  float x1 = (cx - x) * cos(alpha) - (cy - y) * sin(alpha);
  float y1 = (cx - x) * sin(alpha) + (cy - y) * cos(alpha);
  *newVelocity_x = velocity_x * cos(alpha) - velocity_y * sin(alpha);
  *newVelocity_y = velocity_x * sin(alpha) + velocity_y * cos(alpha);
  changeVelocity(x1, y1, radius, -width / 2, height / 2, width / 2, -height / 2, *newVelocity_x, *newVelocity_y, newVelocity_x, newVelocity_y);
  float resultVelocity_x = *newVelocity_x * cos(alpha) + *newVelocity_y * sin(alpha);
  float resultVelocity_y = -*newVelocity_x * sin(alpha) + *newVelocity_y * cos(alpha);
  *newVelocity_x = resultVelocity_x;
  *newVelocity_y = resultVelocity_y;
}

__device__ bool intersectsBallAndRect(float cx, float cy, float radius, float left, float top, float right, float bottom) {
  float closestX = (cx < left ? left : (cx > right ? right : cx));
  float closestY = (cy > top ? top : (cy < bottom ? bottom : cy));
  float dx = closestX - cx;
  float dy = closestY - cy;

  return (dx * dx + dy * dy) <= radius * radius;
}

__device__ bool intersectsBallAndRotatedRect(float cx, float cy, float radius, float x, float y, float width, float height, float angle) {
  float alpha = -angle / 180 * PI;
  float x1 = (cx - x) * cos(alpha) - (cy - y) * sin(alpha);
  float y1 = (cx - x) * sin(alpha) + (cy - y) * cos(alpha);
  return intersectsBallAndRect(x1, y1, radius, -width / 2, height / 2, width / 2, -height / 2);
}

__global__ void CudaResolveBallsCollisionWithPlatformKernel(BallData* ballsData, int size, PlatformData platformData)
{
  int tId = blockIdx.x * blockDim.x + threadIdx.x;
  if (tId < size) {
    if (intersectsBallAndRotatedRect(ballsData[tId].pos_x, ballsData[tId].pos_y, ballsData[tId].radius, platformData.pos_x, platformData.pos_y, platformData.width, platformData.height, platformData.angle)) {
      changeVelocity(ballsData[tId].pos_x, ballsData[tId].pos_y, ballsData[tId].radius, platformData.pos_x, platformData.pos_y, platformData.width, platformData.height, platformData.angle, ballsData[tId].velocity_x, ballsData[tId].velocity_y, &(ballsData[tId].velocity_x), &(ballsData[tId].velocity_y));
    }
  }
}

hipError_t CudaResolveBallsCollisionWithPlatformKernelInvoke(BallData* ballsData, int size, PlatformData platformData)
{
  int threadsPerBlock = 1024;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  CudaResolveBallsCollisionWithPlatformKernel << <blocksPerGrid, threadsPerBlock >> > (ballsData, size, platformData);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    std::cout << "CudaResolveBallsCollisionWithPlatformKernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}

__global__ void CudaResolveBallsCollisionWithBasketKernel(BallData* ballsData, int size, PlatformData platformData, Score* score)
{
  int tId = blockIdx.x * blockDim.x + threadIdx.x;
  if (tId < size) {
    if (intersectsBallAndRotatedRect(ballsData[tId].pos_x, ballsData[tId].pos_y, ballsData[tId].radius, platformData.pos_x, platformData.pos_y, platformData.width, platformData.height, platformData.angle)) {
      if (ballsData[tId].type == RED) {
        score->redCount++;
      } else {
        score->blueCount++;
      }
      ballsData[tId].type = TO_DELETE;
    }
    if (ballsData[tId].pos_x < -1.5f || ballsData[tId].pos_x > 1.5f || ballsData[tId].pos_y < -1.5f || ballsData[tId].pos_y > 1.5f) {
      ballsData[tId].type = TO_DELETE;
    }
  }
}

hipError_t CudaResolveBallsCollisionWithBasketKernelInvoke(BallData* ballsData, int size, PlatformData platformData, Score* score)
{
  int threadsPerBlock = 1024;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  CudaResolveBallsCollisionWithBasketKernel << <blocksPerGrid, threadsPerBlock >> > (ballsData, size, platformData, score);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    std::cout << "CudaResolveBallsCollisionWithBasketKernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
  }

  return cudaStatus;
}


void Ball::CudaMoveBalls(BallData* ballsData, int ballsCount, float deltaTime, PlatformData* platformData, int platformDataCount, PlatformData basketData, Score* score) {
  BallData* ballsDataGPU;
  Score* scoreGPU;

  allocateOnGPU((void**)&ballsDataGPU, ballsCount * sizeof(BallData));
  allocateOnGPU((void**)&scoreGPU, sizeof(Score));
  moveToGPU(ballsData, ballsDataGPU, ballsCount * sizeof(BallData));
  moveToGPU(score, scoreGPU, sizeof(Score));

  CudaMoveBallsKernelInvoke(ballsDataGPU, ballsCount, deltaTime);
  CudaResolveBallsCollisionsKernelInvoke(ballsDataGPU, ballsCount);
  for (int i = 0; i < platformDataCount; i++) {
    CudaResolveBallsCollisionWithPlatformKernelInvoke(ballsDataGPU, ballsCount, platformData[i]);
  }
  CudaResolveBallsCollisionWithBasketKernelInvoke(ballsDataGPU, ballsCount, basketData, scoreGPU);
  moveToCPU(ballsData, ballsDataGPU, ballsCount * sizeof(BallData));
  moveToCPU(score, scoreGPU, sizeof(Score));

  hipFree(ballsDataGPU);
  hipFree(scoreGPU);
}